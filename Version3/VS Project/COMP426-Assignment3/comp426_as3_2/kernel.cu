#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <string>

// OpenGL Graphics includes
#include "GL/glew.h"
#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include "GL/freeglut.h" 
#endif

// Define states for cells
#define HEALTHY  0
#define CANCER   1
#define MEDICINE 2

// 2D area of 1024 x 768 cells
const int g_windowWidth = 1024;
const int g_windowHeight = 768;
int g_quad_read[g_windowWidth][g_windowHeight];
int g_quad_write[g_windowWidth][g_windowHeight];

// Update every 1/30th second
const int g_updateTime = 1.0 / 30.0 * 1000.0;

// At least 25% of cells initialized as cancer cells
const int g_initialCancer = g_windowWidth * g_windowHeight * 0.26;

const int g_font = (int)GLUT_BITMAP_TIMES_ROMAN_24;

hipError_t updateWithCuda();

__global__ void updateKernel(int *devRead, int *devWrite)
{
	/**
	@Desc : Updates each cell state
	@param1 : pointer to read array
	@param2 : pointer to write array
	*/

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (devRead[x*g_windowHeight + y] == HEALTHY || devRead[x*g_windowHeight + y] == CANCER) {
		int _numSurrounded = 0;
		int _before = 0;
		int _after = 0;

		// If a healthy cell is surrounded by >= 6 cancer cells,
		// it becomes a cancer cell
		if (devRead[x*g_windowHeight + y] == HEALTHY) {
			_before = CANCER;
			_after = CANCER;
		}
		// If a cancer cell is surrounded by >= 6 medicine cells,
		// it becomes a healthy cell
		else if (devRead[x*g_windowHeight + y] == CANCER) {
			_before = MEDICINE;
			_after = HEALTHY;
		}

		// Check the states of the surrounding cells
		if (x > 0 && y > 0) {
			if (devRead[(x - 1)*g_windowHeight + (y - 1)] == _before)
				_numSurrounded++;
		}
		if (y > 0) {
			if (devRead[x*g_windowHeight + (y - 1)] == _before)
				_numSurrounded++;
		}
		if (x < (g_windowWidth - 1) && y > 0) {
			if (devRead[(x + 1)*g_windowHeight + (y - 1)] == _before)
				_numSurrounded++;
		}
		if (x > 0) {
			if (devRead[(x - 1)*g_windowHeight + y] == _before)
				_numSurrounded++;
		}
		if (x < (g_windowWidth - 1)) {
			if (devRead[(x + 1)*g_windowHeight + y] == _before)
				_numSurrounded++;
		}
		if (x > 0 && y < (g_windowHeight - 1)) {
			if (devRead[(x - 1)*g_windowHeight + (y + 1)] == _before)
				_numSurrounded++;
		}
		if (y < (g_windowHeight - 1)) {
			if (devRead[x*g_windowHeight + (y + 1)] == _before)
				_numSurrounded++;
		}
		if (x < (g_windowWidth - 1) && y < (g_windowHeight - 1)) {
			if (devRead[(x + 1)*g_windowHeight + (y + 1)] == _before)
				_numSurrounded++;
		}
		// Change state if surrounded by >= 6 of a certain cell
		if (_numSurrounded >= 6) {
			devWrite[x*g_windowHeight + y] = _after;
		}
	}
}

hipError_t updateWithCuda()
{
	/**
	@Desc : Helper function for using CUDA to update cells in parallel. Launches the CUDA kernel
	*/

	int *dev_read = 0;
    int *dev_write = 0;
	std::size_t *pitch_read = new std::size_t;
	std::size_t *pitch_write = new std::size_t;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for arrays
    cudaStatus = hipMallocPitch(&dev_read, pitch_read, g_windowWidth * sizeof(std::size_t), g_windowHeight * sizeof(std::size_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMallocPitch(&dev_write, pitch_write, g_windowWidth * sizeof(std::size_t), g_windowHeight * sizeof(std::size_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy arrays from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_read, g_quad_read, (g_windowWidth*g_windowHeight) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_write, g_quad_write, (g_windowWidth*g_windowHeight) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dim3 dimBlock(16, 32);
	dim3 dimGrid;
	dimGrid.x = (1024 + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (768 + dimBlock.y - 1) / dimBlock.y;

    // Launch a kernel on the GPU with one thread for each element.
    updateKernel<<<dimGrid, dimBlock>>>(dev_read, dev_write);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	// Copy array from GPU buffer to host memory.
    cudaStatus = hipMemcpy(g_quad_write, dev_write, (g_windowWidth*g_windowHeight) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_read);
    hipFree(dev_write);
    
    return cudaStatus;
}

void Update(int value)
{
	/**
	@Desc : Function that uses CUDA to update the cells in parallal, and then calls itself (to update again)
	@param1 : unused parameter that is passed by the glutTimerFunc
	*/

	// Update read array with current data from write array before each new update
	for (int i = 0; i < g_windowWidth; ++i) {
		for (int j = 0; j < g_windowHeight; ++j) {
			g_quad_read[i][j] = g_quad_write[i][j];
		}
	}

	// Update cells in parallel
    hipError_t cudaStatus = updateWithCuda();

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "updateWithCuda failed!");
        return;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return;
    }

	glutPostRedisplay();
	glutTimerFunc(g_updateTime, Update, 0);
}

void RenderBitmapString(float x, float y, void *font, const char *string)
{
	/**
	@Desc : Renders bitmap strings to display text on screen
	@param1 : x position of where text should be displayed
	@param2 : y position of where text should be displayed
	@param3 : font to be used
	@param4 : string text to be displayed on screen
	*/

	const char *c;
	glRasterPos2f(x, y);
	for (c = string; *c != '\0'; c++) {
		glutBitmapCharacter(font, *c);
	}
}

void Display()
{
	/**
	@Desc : Displays the cells and text in a window on screen
	*/

	// Display the cells using OpenGL
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, g_windowWidth, g_windowHeight, 0);

	glClearColor(1, 1, 1, 1);
	glClear(GL_COLOR_BUFFER_BIT);
	glBegin(GL_QUADS);
	int _healthyCount = 0;
	int _cancerCount = 0;
	int _medicineCount = 0;
	for (int x = 0; x < g_windowWidth; x++)
	{
		for (int y = 0; y < g_windowHeight; y++)
		{
			if (g_quad_read[x][y] == HEALTHY)
			{
				// Healthy cells are green
				glColor3f(0, 0.5, 0);
				_healthyCount++;
			}
			else if (g_quad_read[x][y] == CANCER)
			{
				// Cancer cells are red
				glColor3f(1, 0, 0);
				_cancerCount++;
			}
			else if (g_quad_read[x][y] == MEDICINE)
			{
				// Medicine cells are yellow
				glColor3f(1, 1, 0);
				_medicineCount++;
			}
			glVertex2f(x, y);
			glVertex2f(x + 1, y);
			glVertex2f(x + 1, y + 1);
			glVertex2f(x, y + 1);
		}
	}
	glEnd();

	std::string _hCount = std::to_string(static_cast<long long>(_healthyCount));
	const char * _hc = _hCount.c_str();
	std::string _cCount = std::to_string(static_cast<long long>(_cancerCount));
	const char * _cc = _cCount.c_str();
	std::string _mCount = std::to_string(static_cast<long long>(_medicineCount));
	const char * _mc = _mCount.c_str();

	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();
	glLoadIdentity();
	glColor3f(0, 0, 0);
	// Display the number of each type of cell
	RenderBitmapString(0, 30, (void *)g_font, "Healthy: ");
	RenderBitmapString(0, 50, (void *)g_font, _hc);
	RenderBitmapString(0, 100, (void*)g_font, "Cancer: ");
	RenderBitmapString(0, 120, (void *)g_font, _cc);
	RenderBitmapString(0, 170, (void *)g_font, "Medicine: ");
	RenderBitmapString(0, 190, (void *)g_font, _mc);
	glPopMatrix();

	glutSwapBuffers();
}

void Initialize()
{
	/**
	@Desc : Initialization function for glut
	*/

	glMatrixMode(GL_PROJECTION);
	glViewport(0, 0, g_windowWidth, g_windowHeight);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	GLfloat aspect = (GLfloat)g_windowWidth / g_windowHeight;
	gluPerspective(45, aspect, 0.1f, 10.0f);
	glClearColor(0.0, 0.0, 0.0, 0.0);
}

void MouseClicks(int button, int state, int x, int y)
{
	/**
	@Desc : Function that handles mouse buttons being clicked
	@param1 : mouse button that was clicked
	@param2 : state of button that was clicked
	@param3 : x position of pointer when mouse was clicked
	@param4 : y position of pointer when mouse was clicked
	*/

	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) {
		// If medicine is injected on a cancer cell,
		// the medicine is absorbed and the cell turns into a healthy cell
		if (g_quad_read[x][y] == CANCER) {
			g_quad_write[x][y] = HEALTHY;
		}
		// If medicine is injected on a healthy or medicine cell,
		// the medicine is not absorbed and propagates radially outwards by one cell
		else {
			g_quad_write[x][y] = MEDICINE;
			if (x > 0 && y > 0)
				g_quad_write[x - 1][y - 1] = MEDICINE;
			if (y > 0)
				g_quad_write[x][y - 1] = MEDICINE;
			if (x < (g_windowWidth - 1) && y > 0)
				g_quad_write[x + 1][y - 1] = MEDICINE;
			if (x > 0)
				g_quad_write[x - 1][y] = MEDICINE;
			if (x < (g_windowWidth - 1))
				g_quad_write[x + 1][y] = MEDICINE;
			if (x > 0 && y < (g_windowHeight - 1))
				g_quad_write[x - 1][y + 1] = MEDICINE;
			if (y < (g_windowHeight - 1))
				g_quad_write[x][y + 1] = MEDICINE;
			if (x < (g_windowWidth - 1) && y < (g_windowHeight - 1))
				g_quad_write[x + 1][y + 1] = MEDICINE;
		}
	}
}

void Keyboard(unsigned char key, int mousePositionX, int mousePositionY)
{
	/**
	@Desc : Function that handles keyboard buttons being pressed
	@param1 : key that was pressed
	@param2 : x position of mouse pointer
	@param3 : y position of mouse pointer
	*/

	switch (key)
	{
	// Escape key
	case 27:
		exit ( 0 );
		break;

	default:
		break;
	}
}

int main(int argc, char **argv)
{
	/**
	@Desc : Main control thread
	*/

	// initialize
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH );
	glutInitWindowSize(g_windowWidth, g_windowHeight);
	glutCreateWindow("2D Cell Growth Simulation");

	// Initialize all cells as healthy cells
	for (int i = 0; i < 1024; i++)
	{
		for (int j = 0; j < 768; j++)
		{
			g_quad_write[i][j] = HEALTHY;
		}
	}

	// Initialize random seed
	srand(time(NULL));

	// Change at least 25% of cells to cancer cells
	for (int i = 0; i <= g_initialCancer; i++)
	{
		int x = rand() % 1024;
		int y = rand() % 768;
		if (g_quad_write[x][y] == CANCER)
			i--;
		else
			g_quad_write[x][y] = CANCER;
	}

	glutDisplayFunc(Display);
	glutIdleFunc(Display);
	glutMouseFunc(MouseClicks);
	glutKeyboardFunc(Keyboard);
	glutTimerFunc(g_updateTime, Update, 0);
	Initialize();

	glutMainLoop();
	return 0;
}